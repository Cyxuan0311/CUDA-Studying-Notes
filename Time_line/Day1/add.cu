#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <assert.h>

__global__ void vectorAdd(int *a, int *b, int *c, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < size) {
        c[tid] = a[tid] + b[tid];
    }
}

int main() {
    const int size = 1024;
    int *a, *b, *c;
    int *d_a, *d_b, *d_c;

    // 1. 分配并初始化主机内存
    a = (int *)malloc(size * sizeof(int));
    b = (int *)malloc(size * sizeof(int));
    c = (int *)malloc(size * sizeof(int));
    for (int i = 0; i < size; i++) {
        a[i] = i;
        b[i] = 2 * i;
    }

    // 2. 分配设备内存
    hipMalloc(&d_a, size * sizeof(int));
    hipMalloc(&d_b, size * sizeof(int));
    hipMalloc(&d_c, size * sizeof(int));

    // 3. 拷贝数据到设备
    hipMemcpy(d_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size * sizeof(int), hipMemcpyHostToDevice);

    // 4. 启动核函数
    int threadsPerBlock = 256;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, size);

    // 5. 检查核函数执行
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(err));
        return 1;
    }

    // 6. 拷贝结果回主机
    hipMemcpy(c, d_c, size * sizeof(int), hipMemcpyDeviceToHost);

    // 7. 验证结果
    for (int i = 0; i < 5; i++) {
        printf("c[%d] = %d (expected: %d)\n", i, c[i], 3 * i);
        assert(c[i] == 3 * i);  // 验证结果是否正确
    }

    // 8. 释放内存
    free(a); free(b); free(c);
    hipFree(d_a); hipFree(d_b); hipFree(d_c);

    printf("Success!\n");
    return 0;
}